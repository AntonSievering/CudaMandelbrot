#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "MandelBrot.h"
#define OLC_PGE_APPLICATION
#include "olcPixelGameEngine.h"
#include <thread>
#include <fstream>

#define CPI 0.15915494309189

__device__ double cudaSin(double r)
{
	r *= CPI;
	double v = -16 * r * r + 8 * r;
	if (r < 0.5) return v;
	return -v;
}

// z0 = c
// zn+1 = zn * zn + c
__global__ void Mandelbrot(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr - zi * zi + cr;
			zni = 2 * zr * zi + ci;
			zr = znr;
			zi = zni;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

// z0 = c
// zn+1 = zn * zn - c
__global__ void Fractal0(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr - zi * zi - cr;
			zni = 2 * zr * zi - ci;
			zr = znr;
			zi = zni;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

// z0 = c
// zn+1 = zn * zn + 2 * c
__global__ void Fractal1(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr - zi * zi + 2 * cr;
			zni = 2 * zr * zi + 2 * ci;
			zr = znr;
			zi = zni;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

// z0 = c
// zn+1 = zn * zn * zn + c
__global__ void Fractal2(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr * zr - 3 * zr * zi * zi;
			zni = 3 * zr * zr * zi - zi * zi * zi;
			zr = znr + cr;
			zi = zni + ci;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

// z0 = c
// zn+1 = zn * zn * zn * zn + c
__global__ void Fractal3(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr * zr * zr - 6 * zr * zr * zi * zi + zi * zi * zi * zi;
			zni = 4 * zr * zr * zr * zi - 4 * zr * zi * zi * zi;
			zr = znr + cr;
			zi = zni + ci;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

// z0 = c
// zn+1 = zn * zn * zn * zn + c
__global__ void Fractal4(const double fBeginX, const double fBeginY, const double fIncrease, size_t* pIterations, size_t nMaxIterations, double fLimit, int nElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int x = i % 1920;
	int y = i / 1920;

	if (i < nElements)
	{
		double cr, ci, zr, zi, znr, zni;
		size_t nIterations = 0;

		cr = fBeginX + (double)x * fIncrease;
		ci = fBeginY + (double)y * fIncrease;
		zr = cr;
		zi = ci;

		while (zr * zr + zi * zi < fLimit && nIterations < nMaxIterations)
		{
			znr = zr * zr - zi * zi + zr + cr;
			zni = 2 * zr * zi + zi + ci;
			zr = znr + cr;
			zi = zni + ci;

			nIterations++;
		}
		pIterations[i] = nIterations;
	}
}

using namespace std::chrono_literals;

class MandelBrot : public olc::PixelGameEngine
{
private:
	std::vector<olc::Key> vKeys;
	size_t nFractal = 0;

	size_t nMaxIterations = 255;
	olc::Pixel* pPallette = nullptr;
	double fLimit = 1024.0;

	olc::Sprite* sprMandelbrot = nullptr;
	olc::Decal* decMandelbrot = nullptr;
	olc::Sprite* sprXAxis = nullptr;
	olc::Sprite* sprYAxis = nullptr;
	olc::Decal* decXAxis = nullptr;
	olc::Decal* decYAxis = nullptr;

	size_t nThreads = 1;
	std::thread* vThreadPool;
	std::atomic<bool>* bThreadDone;
	bool bRenderStarted = false;
	bool bRendererRestartRequest = false;
	bool bProgrammRunning = true;

	bool bShowAxis = true;
	bool bShowCoords = false;
	bool bSelectionBlocked = false;
	bool bRecording = false;
	size_t nFramesDone = 0;
	std::string sFramesFolderName = "frames";
	size_t nColorMode = 1;

	// pan and zoom
	olc::vd2d panOffset = { 9.44, 5.37 };
	olc::vd2d panStart = olc::vd2d();
	double fZoom = 100.0;

	olc::vi2d vSelectedStart = { -1, -1 };
	olc::vi2d vSelectedSize = { 0, 0 };
	olc::Sprite* sprGrid = nullptr;
	olc::Decal* decGrid = nullptr;

public:
	MandelBrot()
	{
		sAppName = "MandelBrot";
	}

public:
	olc::vd2d WorldToScreen(const olc::vd2d vWorld)
	{
		return (vWorld + panOffset) * fZoom;
	}
	olc::vd2d ScreenToWorld(const olc::vd2d vScreen)
	{
		return vScreen / fZoom - panOffset;
	}

	olc::Pixel GenerateColor(size_t nIterations)
	{
		switch (nColorMode)
		{
		case  1: return GenerateColor1(nIterations); break;
		default: return GenerateColor2(nIterations); break;
		}
	}
	olc::Pixel GenerateColor1(size_t nIterations)
	{
		float a = 0.1f;
		return olc::PixelF(0.5f * sin(a * nIterations) + 0.5f, 0.5f * sin(a * nIterations + 2.094f) + 0.5f, 0.5f * sin(a * nIterations + 4.188f) + 0.5f);
	}
	olc::Pixel GenerateColor2(size_t nIterations)
	{
		int a = 255 - 255 * (double)nIterations / nMaxIterations;
		return olc::Pixel(a, a, a, 255);
	}

	void GeneratePallette()
	{
		olc::Pixel* pPalletteOld = pPallette;
		pPallette = new olc::Pixel[nMaxIterations + 1];

		for (int i = 0; i < nMaxIterations + 1; i++)
		{
			pPallette[i] = GenerateColor(i);
		}

		delete[] pPalletteOld;
	}
	void ClearPixel()
	{
		for (int x = 0; x < 1920; x++)
		{
			for (int y = 0; y < 1080; y++)
			{
				sprMandelbrot->SetPixel(x, y, olc::BLACK);
			}
		}
	}
	void MandelbrotThreadCPU(size_t id, size_t fromX, size_t toX)
	{
		while (true)
		{
			while (bThreadDone[id])
			{
				std::this_thread::sleep_for(1ms);
			}

			for (int x = fromX; x < toX; x++)
			{
				for (int y = 0; y < 1080; y++)
				{
					olc::vd2d world = ScreenToWorld(olc::vi2d(x, y));
					size_t it = itFor(world.x, world.y, nMaxIterations, fLimit);
					sprMandelbrot->SetPixel(x, y, GenerateColor(it));
				}
			}
			bThreadDone[id] = true;
		}
	}
	void MandelbrotThreadCPUIntrinsic(size_t id, size_t fromX, size_t toX)
	{
		while (true)
		{
			while (bThreadDone[id])
			{
				std::this_thread::sleep_for(1ms);
			}

			for (int x = fromX; x < toX; x++)
			{
				for (int y = 0; y < 1080; y += 4)
				{
					double* r = new double[4]; double* i = new double[4];
					for (int j = 0; j < 4; j++)
					{
						auto vWorld = ScreenToWorld(olc::vi2d(x, y + j));
						r[j] = vWorld.x;
						i[j] = vWorld.y;
					}
					
					size_t* it = itForIntrin(r, i, nMaxIterations);

					for (int j = 0; j < 4; j++) sprMandelbrot->SetPixel(x, y, GenerateColor(it[j]));
				}
			}
			bThreadDone[id] = true;
		}
	}
	void MandelbrotThreadCuda(size_t id, size_t fromX, size_t toX)
	{
		GeneratePallette();

		size_t nSize = 1920 * 1080;
		size_t nThreads = 256;
		size_t nBlocks = nSize / nThreads;

		// host memory
		size_t* h_pIterations = (size_t*)malloc(nSize * sizeof(size_t));

		// Device Memory
		size_t* d_pIterations = nullptr;
		// Allocate Memory
		hipMalloc((void**)&d_pIterations, nSize * sizeof(size_t));

		while (bProgrammRunning)
		{
			while (bThreadDone[id])
			{
				std::this_thread::sleep_for(100us);
			}

			olc::vd2d vWorldStart = ScreenToWorld({0.0, 0.0});
			double fWorldInc = ScreenToWorld({ 1, 0 }).x - vWorldStart.x;

			// Start the kernel
			switch (nFractal)
			{
			case 0: Mandelbrot <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize); break;
			case 1: Fractal0 <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize);   break;
			case 2: Fractal1 <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize);   break;
			case 3: Fractal2 <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize);   break;
			case 4: Fractal3 <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize);   break;
			case 5: Fractal4 <<< nBlocks, nThreads >>> (vWorldStart.x, vWorldStart.y, fWorldInc, d_pIterations, nMaxIterations, fLimit, nSize);   break;
			}
			
			hipMemcpy(h_pIterations, d_pIterations, nSize * sizeof(size_t), hipMemcpyDeviceToHost);
			
			for (size_t j = 0; j < nSize; j++)
			{
				size_t x = j % 1920;
				size_t y = j / 1920;
				sprMandelbrot->SetPixel(olc::vi2d(x, y), pPallette[h_pIterations[j]]);
			}

			bThreadDone[id] = true;

			// if in recording mode, save
			if (bRecording)
			{
				auto GenString = [&](int n, int digits)
				{
					std::string str = std::string();
					std::string strn = std::to_string(n);
					int nlen = strn.length();
					for (int i = 0; i < digits - nlen; i++) str.append("0");
					return str + strn;
				};

				std::string sFileName = sFramesFolderName + "/frame" + GenString(nFramesDone, 5) + ".frame";
				std::ofstream file = std::ofstream(sFileName);
				if (file.is_open())
				{
					for (size_t y = 0; y < 1080; y++)
					{
						for (size_t x = 0; x < 1920; x++)
						{
							int idx = y * 1920 + x;
							file << h_pIterations[idx];
							if (x == 1919 && y != 1079) file << "\n";
							else if (x != 1919) file << ",";
						}
					}

					file.close();
				}

				nFramesDone++;
			}
		}

		// free the memory
		free(h_pIterations);
		hipFree(d_pIterations);
	}
	void StartThreads()
	{
		for (int i = 0; i < nThreads; i++)
		{
			bThreadDone[i] = false;
		}
	}
	bool ThreadsDone()
	{
		bool bThreadsDone = true;
		for (int i = 0; i < nThreads; i++)
		{
			bThreadsDone *= bThreadDone[i];
		}
		return bThreadsDone;
	}
	void WaitForThreads()
	{
		while (!ThreadsDone())
		{
			std::this_thread::sleep_for(1ms);
		}
	}

	bool OnUserCreate() override
	{
		sprMandelbrot = new olc::Sprite(1920, 1080);
		decMandelbrot = new olc::Decal(sprMandelbrot);

		vThreadPool = new std::thread[nThreads];
		bThreadDone = new std::atomic<bool>[nThreads];
		size_t pxPerThread = 1920 / nThreads;
		for (int i = 0; i < nThreads; i++)
		{
			vThreadPool[i] = std::thread(&MandelBrot::MandelbrotThreadCuda, this, i, i * pxPerThread, (i + 1) * pxPerThread);
			bThreadDone[i] = false;
		}

		sprXAxis = new olc::Sprite(1920, 1);
		for (int i = 0; i < 1920; i++) sprXAxis->SetPixel(olc::vi2d(i, 0), olc::BLACK);
		decXAxis = new olc::Decal(sprXAxis);
		sprYAxis = new olc::Sprite(1, 1080);
		for (int i = 0; i < 1080; i++) sprYAxis->SetPixel(olc::vi2d(0, i), olc::BLACK);
		decYAxis = new olc::Decal(sprYAxis);

		sprGrid = new olc::Sprite(1920, 1080);
		for (int x = 0; x < 1920; x++)
		{
			for (int y = 0; y < 1080; y++)
			{
				sprGrid->SetPixel(x, y, olc::BLANK);
			}
		}
		decGrid = new olc::Decal(sprGrid);

		vKeys.push_back(olc::Key::K1);
		vKeys.push_back(olc::Key::K2);
		vKeys.push_back(olc::Key::K3);
		vKeys.push_back(olc::Key::K4);
		vKeys.push_back(olc::Key::K5);
		vKeys.push_back(olc::Key::K6);

		return true;
	}
	bool OnUserUpdate(float fElapsedTime) override
	{
		// Draw Hint
		olc::Pixel col;
		olc::vf2d vfScale = { 2.0, 2.0 };
		switch (nColorMode)
		{
		case 0: col = olc::WHITE;
		case 1: col = olc::BLACK;
		}

		auto DrawSpriteLine = [&](olc::Sprite* spr, olc::Pixel col, olc::vi2d pos, int length, int strength, bool bX = true)
		{
			if (length < 0)
			{
				if (bX) pos.x += length;
				else pos.y += length;
				length = -length;
			}

			if (bX)
			{
				for (int x = pos.x; x < pos.x + length; x++)
				{
					for (int y = pos.y; y < pos.y + strength; y++)
					{
						spr->SetPixel({ x, y }, col);
					}
				}
			}
			else
			{
				for (int y = pos.y; y < pos.y + length; y++)
				{
					for (int x = pos.x; x < pos.x + strength; x++)
					{
						spr->SetPixel({ x, y }, col);
					}
				}
			}
		};
		auto DrawSpriteRect = [&](olc::Sprite* spr, olc::Pixel col, olc::vi2d pos, olc::vi2d size, int strength)
		{
			DrawSpriteLine(spr, col, pos, size.x, strength, true);
			DrawSpriteLine(spr, col, pos, size.y, strength, false);
			DrawSpriteLine(spr, col, pos + olc::vi2d(0, size.y), size.x, strength, true);
			DrawSpriteLine(spr, col, pos + olc::vi2d(size.x, 0), size.y, strength, false);
		};

		int i = 0;
		for (auto& key : vKeys)
		{
			if (GetKey(key).bPressed)
			{
				nFractal = i;
				bRendererRestartRequest = true;
			}
			i++;
		}

		Clear(olc::BLACK);
		olc::vi2d mouse = { GetMouseX(), GetMouseY() };

		if (!bRenderStarted)
		{
			StartThreads();
			bRenderStarted = true;
		}

		// Draw the Mandelbrot
		delete decMandelbrot;
		decMandelbrot = new olc::Decal(sprMandelbrot);
		if (ThreadsDone())
		{
			if (bRendererRestartRequest)
			{
				bRendererRestartRequest = false;
				bRenderStarted = false;
			}
		}

		DrawDecal({ 0, 0 }, decMandelbrot);

		if (GetMouse(1).bPressed) panStart = mouse;
		if (GetMouse(1).bHeld)
		{
			panOffset += olc::vd2d((olc::vd2d)(mouse - panStart) / fZoom);
			panStart = mouse;
			bRendererRestartRequest = true;
		}

		if (GetKey(olc::Key::SPACE).bPressed) bShowAxis = !bShowAxis;
		if (bShowAxis)
		{
			olc::vd2d vWorldZero = WorldToScreen({ 0.0, 0.0 });
			DrawDecal(olc::vi2d(vWorldZero.x, 0), decYAxis);
			DrawDecal(olc::vi2d(0, vWorldZero.y), decXAxis);
			DrawStringDecal(olc::vi2d(vWorldZero.x + 10, 10), "Imag [C]", col, { 1.0, 1.0 });
			DrawStringDecal(olc::vi2d(ScreenWidth() - 74, vWorldZero.y - 18), "Real [C]", col, { 1.0, 1.0 });
		}

		// Edit iterations and limits
		if (GetKey(olc::Key::U).bPressed && nMaxIterations > 1)
		{
			nMaxIterations /= 2;
			GeneratePallette();
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::I).bPressed)
		{
			nMaxIterations *= 2;
			GeneratePallette();
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::O).bPressed && nMaxIterations > 1)
		{
			nMaxIterations -= 1;
			GeneratePallette();
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::P).bPressed)
		{
			nMaxIterations += 1;
			GeneratePallette();
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::T).bPressed)
		{
			fLimit /= 2.0;
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::Z).bPressed)
		{
			fLimit *= 2.0;
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::ENTER).bPressed)
		{
			fZoom = 100;
			panOffset = { 9.44, 5.37 };
			panStart = { 0, 0 };
			bRendererRestartRequest = true;
		}

		// Recording
		if (GetKey(olc::Key::F1).bPressed)
		{
			bRecording = !bRecording;
			if (bRecording) bRenderStarted = false;
		}

		// Zoom and pan stuff
		olc::vd2d vMouseBeforeZoom = ScreenToWorld(mouse);
		if (GetKey(olc::Key::Q).bPressed || GetKey(olc::Key::Q).bHeld) fZoom += fZoom * 1.1 * fElapsedTime;
		if (bRecording && !bRenderStarted) fZoom += fZoom * 1.1 / 120.0;
		if (GetKey(olc::Key::E).bPressed || GetKey(olc::Key::E).bHeld) fZoom -= fZoom * 1.1 * fElapsedTime;
		olc::vd2d vMouseAfterZoom = ScreenToWorld(mouse);
		if ((vMouseAfterZoom - vMouseBeforeZoom) != olc::vd2d())
		{
			panOffset += (vMouseAfterZoom - vMouseBeforeZoom);
			bRendererRestartRequest = true;
		}

		// halt if ESC is pressed
		if (GetKey(olc::Key::ESCAPE).bPressed && !GetMouse(0).bHeld) return false;

		// Set Color Mode
		if (GetKey(olc::Key::J).bPressed)
		{
			nColorMode = 0;
			GeneratePallette();
			bRendererRestartRequest = true;
		}
		if (GetKey(olc::Key::K).bPressed)
		{
			nColorMode = 1;
			GeneratePallette();
			bRendererRestartRequest = true;
		}

		if (GetKey(olc::Key::TAB).bPressed) bShowCoords = !bShowCoords;
		if (bShowCoords)
		{
			olc::vd2d vMouseWorld = ScreenToWorld(mouse + olc::vi2d(50, 0));
			olc::Pixel col = olc::BLACK; if (sprMandelbrot->GetPixel(mouse) == olc::BLACK) col = olc::WHITE;
			std::string sCoord = std::to_string(vMouseWorld.x) + " + " + std::to_string(-vMouseWorld.y) + "i";
			DrawStringDecal(mouse, sCoord, col, { 2.0, 2.0 });
		}

		// Draw Selected grid
		if (GetMouse(0).bPressed && !bSelectionBlocked) vSelectedStart = mouse;
		if (GetMouse(0).bHeld && !bSelectionBlocked)
		{
			// Vanish the old grid
			DrawSpriteRect(sprGrid, olc::BLANK, vSelectedStart, vSelectedSize, 2);
			
			// Set the new grid
			vSelectedSize = mouse - vSelectedStart;

			// Draw Grid to the Sprite
			DrawSpriteRect(sprGrid, olc::RED, vSelectedStart, vSelectedSize, 2);

			// Draw Grid to Screen
			delete decGrid;
			decGrid = new olc::Decal(sprGrid);
			SetPixelMode(olc::Pixel::ALPHA);
			DrawDecal({ 0, 0 }, decGrid);
			SetPixelMode(olc::Pixel::NORMAL);
		}
		if (GetMouse(0).bReleased && !bSelectionBlocked)
		{
			DrawSpriteRect(sprGrid, olc::BLANK, vSelectedStart, vSelectedSize, 2);

			if (vSelectedSize.x != 0 && vSelectedSize.y != 0)
			{
				if (vSelectedSize.x < 0)
				{
					vSelectedSize.x = -vSelectedSize.x;
					vSelectedStart.x -= vSelectedSize.x;
				}
				if (vSelectedSize.y < 0)
				{
					vSelectedSize.y = -vSelectedSize.y;
					vSelectedStart.y -= vSelectedSize.y;
				}

				// Zoom in
				panOffset -= (olc::vd2d)(vSelectedStart) / fZoom;
				fZoom *= 1920.0 / (double)vSelectedSize.x;

				bRendererRestartRequest = true;
			}
		}
		if (!GetMouse(0).bHeld && !GetMouse(0).bPressed && !GetMouse(0).bReleased) bSelectionBlocked = false;
		else
		{
			if (GetKey(olc::Key::ESCAPE).bPressed)
			{
				bSelectionBlocked = true;
				DrawSpriteRect(sprGrid, olc::BLANK, vSelectedStart, vSelectedSize, 2);
			}
		}

		DrawStringDecal({ 5, 25 }, "Bedienungshilfe:", col, vfScale);
		DrawStringDecal({ 5, 45 }, "<J> Helligkeitsstufen", col, vfScale);
		DrawStringDecal({ 5, 65 }, "<K> Bunte Farben", col, vfScale);
		DrawStringDecal({ 5, 85 }, "<ESC> Beenden", col, vfScale);
		DrawStringDecal({ 5, 105 }, "<Q> Hineinzoomen", col, vfScale);
		DrawStringDecal({ 5, 125 }, "<E> Hinauszoomen", col, vfScale);
		std::string s = bShowAxis ? "verstecken" : "zeigen";
		DrawStringDecal({ 5, 145 }, "<SPACE> Koordinatensystem " + s, col, vfScale);
		s = bShowCoords ? "verstecken" : "zeigen";
		DrawStringDecal({ 5, 165 }, "<TAB> Korrdinate " + s, col, vfScale);
		DrawStringDecal({ 5, 185 }, "<U> maximale Interationen halbieren", col, vfScale);
		DrawStringDecal({ 5, 205 }, "<I> maximale Iterationen verdoppeln", col, vfScale);
		DrawStringDecal({ 5, 225 }, "<O> maximale Iterationen um 1 inkrementieren", col, vfScale);
		DrawStringDecal({ 5, 245 }, "<P> maximale Iterationen um 1 dekrementieren", col, vfScale);
		DrawStringDecal({ 5, 265 }, "<T> Limit halbieren", col, vfScale);
		DrawStringDecal({ 5, 285 }, "<Z> Limit verdoppeln", col, vfScale);
		DrawStringDecal({ 5, 305 }, "<ENTER> Standart-Zoom", col, vfScale);

		DrawStringDecal({ 5, 345 }, "aktuelles Limit:" + std::to_string(fLimit), col, vfScale);
		DrawStringDecal({ 5, 365 }, "aktuelle maximale Iterationen: " + std::to_string(nMaxIterations), col, vfScale);
		DrawStringDecal({ 5, 385 }, "Zoom: " + std::to_string(fZoom), col, vfScale);
		DrawStringDecal({ 5, 405 }, "Du nimmst gerade " + (std::string)(bRecording ? "" : "nicht ") + "auf", col, vfScale);

		DrawStringDecal({ 1500, 25 }, "aufgenommene Frames: " + std::to_string(nFramesDone), col, vfScale);

		return true;
	}
	bool OnUserDestroy() override
	{
		bProgrammRunning = true;
		for (int i = 0; i < nThreads; i++)
		{
			vThreadPool[i].detach();
		}
		return true;
	}
};

int main()
{
	MandelBrot brot;
	if (brot.Construct(1920, 1080, 1, 1, true))
		brot.Start();
	return 0;
}